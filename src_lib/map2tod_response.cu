#include "hip/hip_runtime.h"
#include "../include/gpu_mm.hpp"
#include "../include/plan_iterator.hpp"

#include <cassert>
#include <ksgpu/cuda_utils.hpp>

using namespace ksgpu;

namespace gpu_mm {
#if 0
}   // pacify editor auto-indent
#endif


// The "pre-map2tod" kernel partially zeroes the TOD.
// Launch with threadIdx = { 32*W } (not {32,W}).

template<typename T>
__global__ void pre_map2tod_kernel(T *tod, const ulong *plan_mt, uint nmt, uint nmt_per_block)
{
    uint imt0 = (blockIdx.x) * nmt_per_block + threadIdx.x;
    uint imt1 = (blockIdx.x + 1) * nmt_per_block;
    imt1 = (imt1 < nmt) ? imt1 : nmt;
    imt1 = (imt1 + 31U) & ~31U;

    for (uint imt = imt0; imt < imt1; imt += blockDim.x) {
        uint i = (imt < nmt) ? imt : (nmt-1);
        ulong mt = plan_mt[i];

        uint icl_flagged = uint(mt >> 20);
        uint icl = icl_flagged & ((1U << 26) - 1);
        bool zflag = (icl_flagged & (1U << 27)) != 0;
        uint mask = __ballot_sync(ALL_LANES, (imt < nmt) && zflag);

        for (uint lane = 0; lane < 32; lane++) {
            if (mask & (1U << lane)) {
                uint zcl = __shfl_sync(ALL_LANES, icl, lane);
                uint s = (ulong(zcl) << 5) + (threadIdx.x & 31);
                tod[s] = 0;
            }
        }
    }
}


template<typename T>
static void launch_pre_map2tod(T *tod, const ulong *plan_mt, int nmt)
{
    static constexpr int W = 4;  // warps per threadblock
    static constexpr int nmt_per_block = 1024;

    xassert(tod != nullptr);
    xassert(plan_mt != nullptr);
    xassert(nmt > 0);

    int nblocks = (nmt + nmt_per_block - 1) / nmt_per_block;

    pre_map2tod_kernel<T> <<< nblocks, 32*W >>>
        (tod, plan_mt, nmt, nmt_per_block);

    CUDA_PEEK("pre_map2tod kernel launch");
}


// -------------------------------------------------------------------------------------------------


// Helper for map2tod_kernel().
template<typename T>
__device__ T eval_tqu(T *sp, int iy, int ix, T t_resp, T q_resp, T u_resp)
{
    bool in_cell = ((ix | iy) & ~63) == 0;
    int s = (iy << 6) | ix;

    T ret = in_cell ? (t_resp*sp[s] + q_resp*sp[s+64*64] + u_resp*sp[s+2*64*64]) : 0;
    __syncwarp();
    return ret;
}


template<typename T, int W, bool Debug>
__global__ void __launch_bounds__(32*W, 1)
response_map2tod_kernel(
    T *tod,
    const T *lmap,
    const T *xpointing,
    const T *response, // [{t,p},ndet] flattened
    const long *cell_offsets,
    const ulong *plan_mt,
    uint *errflags,
    long ndet,
    long nperdet,
    int nypix_global,
    int nxpix_global,
    int nycells,
    int nxcells,
    long ystride,
    long polstride,
    uint nmt,
    uint nmt_per_block,
    bool periodic_xcoord,
    bool partial_pixelization)
{
    // 48 KB in single precision, 96 KB in double precision.
    // __shared__ T shmem[3*64*64];
    T *shmem = dtype<T>::get_shmem();

    if constexpr (Debug) {
        assert(blockDim.x == 32);
        assert(blockDim.y == W);
    }

    // Threadblock has shape (32,W), so threadIdx.x is the laneId, and threadIdx.y is the warpId.
    const uint laneId = threadIdx.x;
    const uint warpId = threadIdx.y;
    uint err = 0;

    plan_iterator<W,Debug> iterator(plan_mt, nmt, nmt_per_block);
    pixel_locator<T> px(nypix_global, nxpix_global, periodic_xcoord);

    long nsamp = ndet*nperdet;

    // Outer loop over map cells

    while (iterator.get_cell()) {
        uint icell = iterator.icell;
        uint iycell = icell >> 10;
        uint ixcell = icell & ((1<<10) - 1);

        bool valid = (iycell < nycells) && (ixcell < nxcells);
        long offset = valid ? cell_offsets[iycell*nxcells + ixcell] : -1;
        err = ((offset >= 0) || partial_pixelization) ? err : errflag_not_in_pixelization;

        // Global -> shared

        if (offset >= 0) {
            for (int y = warpId; y < 64; y += W) {
                for (int x = laneId; x < 64; x += 32) {
                    int ss = 64*y + x;                 // shared memory offset
                    long sg = offset + y*ystride + x;  // global memory offset

                    shmem[ss] = lmap[sg];
                    shmem[ss + 64*64] = lmap[sg + polstride];
                    shmem[ss + 2*64*64] = lmap[sg + 2*polstride];
                }
            }
            __syncthreads();
        }

        // Inner loop over TOD cache lines

        while (iterator.get_cl()) {
            bool mflag = iterator.icl_flagged & (1U << 26);
            uint icl = iterator.icl;
            long s   = (long(icl) << 5) + laneId;

            if (offset < 0) {
                if (!mflag)
                    tod[s] = 0;
                continue;
            }

            T ypix = xpointing[s];
            T xpix = xpointing[s + nsamp];
            T alpha = xpointing[s + 2*nsamp];

            // T response
            long det = s/nperdet;
            T t_resp = response[det];
            // Q and U response
            T p_resp = response[det+ndet];
            T cos_2a, sin_2a;
            dtype<T>::xsincos(2*alpha, &sin_2a, &cos_2a);
            T q_resp = p_resp*cos_2a;
            T u_resp = p_resp*sin_2a;

            // Locate pixel in shared memory.
            px.locate(ypix, xpix, iycell, ixcell, err);

            // Interpolate local map in shared memory.
            // Note: eval_tqu() returns zero if pixel access is outside current map cell.
            T t = (1-px.dy) * (1-px.dx) * eval_tqu(shmem, px.iy0, px.ix0, t_resp, q_resp, u_resp);
            t +=  (1-px.dy) *   (px.dx) * eval_tqu(shmem, px.iy0, px.ix1, t_resp, q_resp, u_resp);
            t +=    (px.dy) * (1-px.dx) * eval_tqu(shmem, px.iy1, px.ix0, t_resp, q_resp, u_resp);
            t +=    (px.dy) *   (px.dx) * eval_tqu(shmem, px.iy1, px.ix1, t_resp, q_resp, u_resp);

            if (mflag)
                atomicAdd(tod+s, t);
            else
                tod[s] = t;
        }

        __syncthreads();
    }

    // No need for __syncthreads() before write_errflags(), since main loop has __syncthreads() at bottom.
    // Reminder: write_errflags() assumes thread layout is {32,W,1}, and block layout is {B,1,1}.
    write_errflags(errflags, (uint *)shmem, err);
}


template<typename T>
void launch_response_map2tod(
    ksgpu::Array<T> &tod,                       // shape (nsamp,) or (ndet,nt)
    const ksgpu::Array<T> &local_map,           // total size (3 * local_pixelization.npix)
    const ksgpu::Array<T> &xpointing,           // shape (3,nsamp) or (3,ndet,nt)    where axis 0 = {y,x,alpha}
    const ksgpu::Array<T> &response,            // shape (2,ndet)
    const LocalPixelization &local_pixelization, 
    const PointingPlan &plan,
    bool partial_pixelization,
    bool debug)
{
    static constexpr int W = 16;  // warps per threadblock
    static constexpr int shmem_nbytes = 3 * 64 * 64 * sizeof(T);

    check_tod(tod, plan.nsamp, "launch_response_map2tod", true);                      // on_gpu = true
    check_local_map(local_map, local_pixelization, "launch_response_map2tod", true);  // on_gpu = true
    check_xpointing(xpointing, plan.nsamp, "launch_response_map2tod", true);          // on_gpu = true
    long ndet, nperdet;
    check_response(response, plan.nsamp, ndet, nperdet, "launch_response_map2tod", true); // on_gpu = true

    // Verify consistency of (nypix, nxpix, periodic_xcoord) between plan and lppix
    xassert_eq(local_pixelization.nypix_global, plan.nypix_global);
    xassert_eq(local_pixelization.nxpix_global, plan.nxpix_global);
    xassert_eq(local_pixelization.periodic_xcoord, plan.periodic_xcoord);

    launch_pre_map2tod(tod.data, plan.plan_mt, plan.pp.plan_nmt);

    if (debug) {
        response_map2tod_kernel<T,W,true> <<< plan.pp.pointing_nblocks, {32,W}, shmem_nbytes >>>
            (tod.data,                                  // T *tod
             local_map.data,                            // const T *lmap
             xpointing.data,                            // const T *xpointing
             response.data,                             // const T *response
             local_pixelization.cell_offsets_gpu.data,  // const long *cell_offsets
             plan.plan_mt,                              // const ulong *plan_mt
             plan.err_gpu,                              // uint *errflags
             ndet,                                      // long ndet (nsamp=ndet*nperdet)
             nperdet,                                   // long nperdet
             plan.nypix_global,                         // int nypix_global
             plan.nxpix_global,                         // int nxpix_global
             local_pixelization.nycells,                // int nycells
             local_pixelization.nxcells,                // int nxcells
             local_pixelization.ystride,                // long ystride
             local_pixelization.polstride,              // long polstride
             plan.pp.plan_nmt,                          // uint nmt
             plan.pp.nmt_per_threadblock,               // uint nmt_per_block,
             plan.periodic_xcoord,                      // bool periodic_xcoord
             partial_pixelization);                     // bool partial_pixelization
    }
    else {
        response_map2tod_kernel<T,W,false> <<< plan.pp.pointing_nblocks, {32,W}, shmem_nbytes >>>
            (tod.data,                                  // T *tod
             local_map.data,                            // const T *lmap
             xpointing.data,                            // const T *xpointing
             response.data,                             // const T *response
             local_pixelization.cell_offsets_gpu.data,  // const long *cell_offsets
             plan.plan_mt,                              // const ulong *plan_mt
             plan.err_gpu,                              // uint *errflags
             ndet,                                      // long ndet (nsamp=ndet*nperdet)
             nperdet,                                   // long nperdet
             plan.nypix_global,                         // int nypix_global
             plan.nxpix_global,                         // int nxpix_global
             local_pixelization.nycells,                // int nycells
             local_pixelization.nxcells,                // int nxcells
             local_pixelization.ystride,                // long ystride
             local_pixelization.polstride,              // long polstride
             plan.pp.plan_nmt,                          // uint nmt
             plan.pp.nmt_per_threadblock,               // uint nmt_per_block,
             plan.periodic_xcoord,                      // bool periodic_xcoord
             partial_pixelization);                     // bool partial_pixelization
    }

    CUDA_PEEK("map2tod kernel launch");

    // FIXME check_gpu_errflags() causes measurable slowdown (0.5 ms/call).
    // This isn't large enough to be a high priority, but I'd like to revisit it at
    // some point, mostly for the sake of my own understanding. (I don't understand
    // why it would slow things down so much!)

    uint errflags_to_ignore = partial_pixelization ? errflag_not_in_pixelization : 0;
    check_gpu_errflags(plan.err_gpu, plan.pp.pointing_nblocks, "map2tod", errflags_to_ignore);
}


#define INSTANTIATE(T) \
    template void launch_response_map2tod( \
        ksgpu::Array<T> &tod,        \
        const ksgpu::Array<T> &local_map, \
        const ksgpu::Array<T> &xpointing, \
        const ksgpu::Array<T> &response, \
        const LocalPixelization &local_pixelization, \
        const PointingPlan &plan, \
        bool partial_pixelization, \
        bool debug)

INSTANTIATE(float);
INSTANTIATE(double);


}  // namespace gpu_mm
