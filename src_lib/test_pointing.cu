//nvcc -o libtest_pointing.so test_pointing.cu -shared -lcublas -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "hipblas.h"



__global__
void  fillA(float *dra_bore, float *ddec_bore, int n,float *dA)
{
  //int ra_ord=4;
  //int dec_ord=3;
  //int t_ord=2;
  int myi_off=blockIdx.x*blockDim.x+threadIdx.x;
  int nthread=blockDim.x*gridDim.x;
  for (int myind=myi_off;myind<n;myind+=nthread) {
    float myra=dra_bore[myind];
    float mydec=ddec_bore[myind];
    float tmp=myra*myra;

    dA[myind]=1.0;
    float t=2.0*(myind-(n-1))/(n-1.0)+1.0;
    dA[myind+1*n]=t;
    dA[myind+2*n]=t*t;


    dA[myind+6*n]=myra;
    dA[myind+7*n]=tmp;
    dA[myind+8*n]=myra*tmp;
    dA[myind+9*n]=tmp*tmp;
    tmp=mydec*mydec;
    dA[myind+3*n]=mydec;
    dA[myind+4*n]=tmp;
    dA[myind+5*n]=tmp*mydec;
    dA[myind+10*n]=myra*t;
    dA[myind+11*n]=mydec*t;
    dA[myind+12*n]=mydec*myra;
  }
}

/*--------------------------------------------------------------------------------*/
extern "C" {
void eval_fit(float *out,float *fitp, int n, int ndet, float *ra_bore, float *dec_bore)
{
  float *dA;
  int npar=13; //because we're super fragile right now
  if (hipMalloc((void **)&dA,sizeof(float)*n*npar)!=hipSuccess)
    fprintf(stderr,"error in cudaMalloc\n");
  float *dra_bore;
  if (hipMalloc((void **)&dra_bore,sizeof(float)*n*npar)!=hipSuccess)
    fprintf(stderr,"error in cudaMalloc\n");

  float *ddec_bore;
  if (hipMalloc((void **)&ddec_bore,sizeof(float)*n*npar)!=hipSuccess)
    fprintf(stderr,"error in cudaMalloc\n");

  float *dfitp;
  if (hipMalloc((void **)&dfitp,sizeof(float)*ndet*npar)!=hipSuccess)
    fprintf(stderr,"error in cudaMalloc\n");  
  if (hipMemcpy(dfitp,fitp,ndet*npar*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (hipMemcpy(dra_bore,ra_bore,n*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying ra bore data to device.\n");
  if (hipMemcpy(ddec_bore,dec_bore,n*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying dec bore data to device.\n");


  float *dout;
  if (hipMalloc((void **)&dout,sizeof(float)*ndet*n)!=hipSuccess)
    fprintf(stderr,"error in cudaMalloc\n");  

  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat=hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return;
  }
  float one=1.0;
  float zero=0.0;

  hipDeviceSynchronize();
  for (int i=0;i<10;i++) {
    double t1=omp_get_wtime();
    fillA<<<128,128>>>(dra_bore,ddec_bore,n,dA);
    stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,n,ndet,npar,&one,dA,n,dfitp,ndet,&zero,dout,n);
    if (stat!=HIPBLAS_STATUS_SUCCESS) 
      printf("Error in sgemm.\n");
    hipDeviceSynchronize();
    double t2=omp_get_wtime(); 
    printf("Pointing reconstruction took %12.4g\n",t2-t1);
  }
  if (hipMemcpy(out,dout,n*ndet*sizeof(float),hipMemcpyDeviceToHost)!=hipSuccess)
    fprintf(stderr,"Error copying out back to host.\n");
  hipFree(dA);
  hipFree(dra_bore);
  hipFree(ddec_bore);
  hipFree(dfitp);
}
}